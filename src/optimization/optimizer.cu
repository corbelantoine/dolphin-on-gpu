#include "hip/hip_runtime.h"
#include "optimizer.cuh"


namespace opt
{

__constant__ fin::Asset* all_assets;
__constant__ int* portfolio_assets;

void print_cov(float* cov, int n = 20) {
  for (int i = 0; i < n; ++i){
    for (int j = 0; j < n; ++j)
      std::cout <<cov[i * n + j] << " ";
    std::cout << std::endl;
  }
}

__host__ void check_error(hipError_t err)
{
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err)
    << "in " << __FILE__
    << "at line " << __LINE__
    << std::endl;
    exit(EXIT_FAILURE);
  }
}

__device__ void optimize_portfolio(fin::Portfolio& p, hlp::Date& d1, hlp::Date& d2, int verbose)
{
  Workspace work;
  Settings settings;
  Params params;
  Vars vars;

  set_defaults(settings);
  setup_indexing(work, vars);

  const int p_size = 20;

  // setting the quadratic problem
  // set Sigma to the covariance matrix
  float* cov = p.get_covariance(d1, d2);
  int cov_size = p_size * p_size;
  for (int i = 0; i < cov_size; ++i)
    params.Sigma[i] = cov[i];

  // set Returns to returns
  float* returns = p.get_returns(d1, d2);
  for (int i = 0; i < p_size; ++i)
    params.Returns[i] = returns[i];

  params.lambda[0] = 0.8;

  // Solve problem
  settings.verbose = verbose;
  solve(work, settings, params, vars);

  // get solution (optimal weights)
  float weights[p_size];
  for (int i = 0; i < p_size; ++i)
    weights[i] = vars.Weights[i];

  // set portfolio weights
  p.set_weights(weights);

  // free cov and ret
  delete[] cov;
  delete[] returns;
}

__global__ void optimize_portfolios_kernel(fin::Portfolio* d_portfolios, float* d_sharp,
                                hlp::Date& d1, hlp::Date& d2,
                                const int nb_p, const int p_size)
{
  // get portfolio index
  int portfolio_idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (portfolio_idx < nb_p) {
    // create portfolio
    const int size = 20;
    fin::Portfolio p = fin::Portfolio();
    fin::Asset* p_assets[size];
    float p_weights[size];
    for (int j = 0; j < p_size; ++j) {
      // portfolio_assets is a global __constant__
      int asset_id = portfolio_assets[portfolio_idx * p_size + j];
      p_weights[j] = 1. / p_size;
      p_assets[j] = &all_assets[asset_id];
    }
    // set portfolio assets and weights
    p.set_assets(p_assets);
    p.set_weights(p_weights);
    // optimize portfolio (get optimal weights)
    optimize_portfolio(p, d1, d2, 0);
    // save portfolio to shared memory
    d_portfolios[portfolio_idx] = p;
    // set portfolio sharp for further use
    d_sharp[portfolio_idx] = p.get_sharp(d1, d2);
  }
}

__host__ fin::Portfolio get_optimal_portfolio_gpu(fin::Asset *h_assets, int *map_portfolio_assets,
                                    hlp::Date& d1, hlp::Date& d2,
                                    const int nb_assets, const int nb_p, const int p_size)
{
  fin::Portfolio h_portfolios[nb_p];
  fin::Portfolio *d_portfolios;

  float h_sharp[nb_p];
  float* d_sharp;

  fin::Portfolio optimal_portfolio(p_size);

  // copy values to cuda constants (cpu to gpu)
  hipMemcpyToSymbol(HIP_SYMBOL(all_assets), h_assets, sizeof(fin::Asset) * nb_assets);
  hipMemcpyToSymbol(HIP_SYMBOL(portfolio_assets), map_portfolio_assets, sizeof(int) * nb_p * p_size);
  // cuda malloc device portfolios and sharps
  hipError_t err = hipMalloc((void **) &d_portfolios, sizeof(fin::Portfolio) * nb_p);
  check_error(err);
  err = hipMalloc((void **) &d_sharp, sizeof(float) * nb_p);
  check_error(err);

  // TODO adapt grid and block size
  dim3 DimGrid(((nb_assets - 1) / 256, 1, 1));
  dim3 DimBlock(256, 1, 1);
  optimize_portfolios_kernel<<<DimGrid, DimBlock>>>(d_portfolios, d_sharp,
                                                    d1, d2, nb_p, p_size);

  // copy optimized portfolios and their sharp values from gpu to cpu
  hipMemcpy(h_portfolios, d_portfolios, sizeof(fin::Portfolio) * nb_p, hipMemcpyDeviceToHost);
  hipMemcpy(h_sharp, d_sharp, sizeof(float) * nb_p, hipMemcpyDeviceToHost);
  // free cuda memory
  hipFree(d_portfolios);
  hipFree(d_sharp);

  // get portfolio with max sharp
  int max_idx = 0;
  float max_sharp = h_sharp[0];
  for (int i = 0; i < nb_p; ++i) {
    if (h_sharp[i] > max_sharp) {
      max_sharp = h_sharp[i];
      max_idx = i;
    }
  }
  // set optimal portfolio
  optimal_portfolio = h_portfolios[max_idx];

  // free host memory
  delete[] h_portfolios;
  delete[] h_sharp;

  return optimal_portfolio;
}

__host__ fin::Portfolio get_optimal_portfolio_cpu(fin::Asset *h_assets, int *map_portfolio_assets,
                                    hlp::Date& d1, hlp::Date& d2,
                                    const int nb_assets, const int nb_p, const int p_size)
{
  fin::Portfolio optimal_portfolio(p_size);
  float max_sharp = 0;

  // optimize portfolios and return the one with max sharp
  for (int i = 0; i < nb_p; ++i) {
    // create portfolio
    fin::Portfolio p = fin::Portfolio(p_size);
    // declare portfolio assets and weights
    fin::Asset* p_assets[p_size];
    float p_weights[p_size];
    for (int j = 0; j < p_size; ++j) {
      // get portfolio assets and weights
      int asset_id = map_portfolio_assets[i * p_size + j];
      p_weights[j] = 1. / p_size;
      p_assets[j] = &h_assets[asset_id];
    }
    // set portfolio assets and weights
    p.set_assets(p_assets);
    p.set_weights(p_weights);
    // optimize portfolio (get optimal weights)
    optimize_portfolio(p, d1, d2, 0);
    // get portfolio with max sharp
    float sharp = p.get_sharp(d1, d2);
    if (sharp >= max_sharp) {
      max_sharp = sharp;
      optimal_portfolio = p;
    }
  }

  return optimal_portfolio;
}

}
